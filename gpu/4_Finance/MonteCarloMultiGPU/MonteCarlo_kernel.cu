#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

////////////////////////////////////////////////////////////////////////////////
// Global types
////////////////////////////////////////////////////////////////////////////////
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo_common.h"

////////////////////////////////////////////////////////////////////////////////
// Helper reduction template
// Please see the "reduction" CUDA Sample for more information
////////////////////////////////////////////////////////////////////////////////
#include "MonteCarlo_reduction.cuh"

////////////////////////////////////////////////////////////////////////////////
// Internal GPU-side data structures
////////////////////////////////////////////////////////////////////////////////
#define MAX_OPTIONS (1024*1024)

#define THREAD_N         (__TEST_S__)



#ifndef __TEST_MAX_CPU__
#error "No define of __TEST_MAX_CPU__"
#endif

#define MAX_CPU  (__TEST_MAX_CPU__)
#define STATE_SHARE (__TEST_STATE_SHARE__)
#define THUNDERING_TEST (__TEST_THUNDERING_TEST__)

//Preprocessed input option data
typedef struct
{
    real S;
    real X;
    real MuByT;
    real VBySqrtT;
} __TOptionData;

////////////////////////////////////////////////////////////////////////////////
// Overloaded shortcut payoff functions for different precision modes
////////////////////////////////////////////////////////////////////////////////
__device__ inline float endCallValue(float S, float X, float r, float MuByT, float VBySqrtT)
{
    float callValue = S * __expf(MuByT + VBySqrtT * r) - X;
    return (callValue > 0.0F) ? callValue : 0.0F;
}

__device__ inline double endCallValue(double S, double X, double r, double MuByT, double VBySqrtT)
{
    double callValue = S * exp(MuByT + VBySqrtT * r) - X;
    return (callValue > 0.0) ? callValue : 0.0;
}


typedef uint32_t                rng32_t;
typedef uint64_t                rng_state_t;

#define RNG_INC             (0x000000000000006dULL)
#define RNG_A               (0x5851f42d4c957f2dULL)


__device__ inline uint32_t next0(uint64_t *s_) {

    uint64_t s1 = s_[ 0 ];
    const uint64_t s0 = s_[ 1 ];
    s_[ 0 ] = s0;
    s1 ^= s1 << 23;
    return ( s_[ 1 ] = ( s1 ^ s0 ^ ( s1 >> 17 ) ^ ( s0 >> 26 ) ) );
    /*
    uint32_t *s = (uint32_t * )s_;
    uint32_t &x = s[0], &y = s[1], &z = s[2], &w = s[3];

    uint32_t t = x ^ (x << 11);

    x = y; y = z; z = w;
    w = w ^ (w >> 19) ^ (t ^ (t >> 8));
    return z;
     */
}

__device__ inline rng32_t pcg_output_xsh_rs_64_32(rng_state_t state)
{

    return (rng32_t)(((state >> 22u) ^ state) >> ((state >> 61u) + 22u));
}


__device__ inline rng32_t  output_function(rng_state_t oldstate)
{
    rng32_t xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
    rng32_t rot = oldstate >> 59u;
    return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}


__device__ inline rng_state_t multi_stream(rng_state_t state,  const int  seq)
{
    return state + ((RNG_A * (seq << 1u)) | 1u);
}


////////////////////////////////////////////////////////////////////////////////
// This kernel computes the integral over all paths using a single thread block
// per option. It is fastest when the number of thread blocks times the work per
// block is high enough to keep the GPU busy.
////////////////////////////////////////////////////////////////////////////////
static __global__ void MonteCarloOneBlockPerOption(
    hiprandStatePhilox4_32_10_t * __restrict rngStates,
    const __TOptionData * __restrict d_OptionData,
    __TOptionValue * __restrict d_CallValue,
    int pathN,
    int optionN)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    const int SUM_N = THREAD_N;
    __shared__ real s_SumCall[SUM_N];
    __shared__ real s_Sum2Call[SUM_N];

    __shared__ rng_state_t  state[SUM_N];


    // determine global thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Copy random number state to local memory for efficiency
    hiprandStatePhilox4_32_10_t localState = rngStates[tid]; // common for thundering and cuda

    __shared__ rng_state_t share_state[2];
    // = {
    //    0x185706b82c2e03f8ULL,
    //    0x2b47fed88766bb05ULL
    //}
     __shared__ uint64_t  dstate[SUM_N][2];



    share_state[0] = 0x185706b82c2e03f8ULL;
    share_state[1] = 0x2b47fed88766bb05ULL;

    for (int optionIndex = blockIdx.x; optionIndex < optionN; optionIndex += gridDim.x)
    {
        unsigned int count = 0;
        unsigned int sum =  0;


        for (int iSum = threadIdx.x; iSum < SUM_N; iSum += blockDim.x)
        {
            state[iSum] = 0x185706b82c2e03f8ULL + tid;
            dstate[iSum][0] = ((uint64_t *)(& localState))[0];
            dstate[iSum][1] = ((uint64_t *)(& localState))[1];
#pragma unroll 8
            for (int i = iSum; i < pathN; i += SUM_N)
            {

#if THUNDERING_TEST
#if STATE_SHARE

                rng_state_t oldstate = share_state[count & 0x01];
                if (i == 0)
                {
                    share_state[(count + 1) & 0x01] = oldstate * RNG_A + ((tid << 1) | 0x01);
                }
                cg::sync(cta);
                rng_state_t updated_state = multi_stream(share_state[(count + 1) & 0x01], tid);
                count ++;
#else

                rng_state_t oldstate = state[iSum];
                state[iSum] = oldstate * RNG_A + ((tid << 1) | 0x01);
                rng_state_t updated_state = state[iSum];
#endif

                //rng_state_t updated_state = oldstate;

                uint32_t d = next0(dstate[iSum]);
                //uint32_t d = 1;
                unsigned int          r = output_function(updated_state)^ d ;
#else

                unsigned int          r = hiprand(&localState);
#endif
                sum += r;
            }

            s_SumCall[iSum]  = sum;
        }

        //Reduce shared memory accumulators
        //and write final result to global memory
        cg::sync(cta);
        sumReduce<real, SUM_N, THREAD_N>(s_SumCall, s_Sum2Call, cta, tile32, &d_CallValue[optionIndex]);
    }
}

static __global__ void rngSetupStates(
    hiprandStatePhilox4_32_10_t *rngState,
    int device_id)
{
    // determine global thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each threadblock gets different seed,
    // Threads within a threadblock get different sequence numbers
    hiprand_init(blockIdx.x + gridDim.x * device_id, threadIdx.x, 0, &rngState[tid]);
}



////////////////////////////////////////////////////////////////////////////////
// Host-side interface to GPU Monte Carlo
////////////////////////////////////////////////////////////////////////////////
extern "C" void initMonteCarloGPU(TOptionPlan *plan)
{
    checkCudaErrors(hipMalloc(&plan->d_OptionData, sizeof(__TOptionData) * (plan->optionCount)));
    checkCudaErrors(hipMalloc(&plan->d_CallValue, sizeof(__TOptionValue) * (plan->optionCount)));
    checkCudaErrors(hipHostMalloc(&plan->h_OptionData, sizeof(__TOptionData) * (plan->optionCount)));
    //Allocate internal device memory
    checkCudaErrors(hipHostMalloc(&plan->h_CallValue, sizeof(__TOptionValue) * (plan->optionCount)));
    //Allocate states for pseudo random number generators
    checkCudaErrors(hipMalloc((void **) &plan->rngStates,
                               plan->gridSize * THREAD_N * sizeof(hiprandStatePhilox4_32_10_t)));
    checkCudaErrors(hipMemset(plan->rngStates, 0, plan->gridSize * THREAD_N * sizeof(hiprandStatePhilox4_32_10_t)));

    // place each device pathN random numbers apart on the random number sequence
    rngSetupStates <<< plan->gridSize, THREAD_N >>> (plan->rngStates, plan->device);
    getLastCudaError("rngSetupStates kernel failed.\n");
}

//Compute statistics and deallocate internal device memory
extern "C" void closeMonteCarloGPU(TOptionPlan *plan)
{
    for (int i = 0; i < plan->optionCount; i++)
    {
        const double    RT = plan->optionData[i].R * plan->optionData[i].T;
        const double   sum = plan->h_CallValue[i].Expected;
        const double  sum2 = plan->h_CallValue[i].Confidence;
        const double pathN = plan->pathN;
        //Derive average from the total sum and discount by riskfree rate
        plan->callValue[i].Expected = (float)(exp(-RT) * sum / pathN);
        //Standard deviation
        double stdDev = sqrt((pathN * sum2 - sum * sum) / (pathN * (pathN - 1)));
        //Confidence width; in 95% of all cases theoretical value lies within these borders
        plan->callValue[i].Confidence = (float)(exp(-RT) * 1.96 * stdDev / sqrt(pathN));
    }

    checkCudaErrors(hipFree(plan->rngStates));
    checkCudaErrors(hipHostFree(plan->h_CallValue));
    checkCudaErrors(hipHostFree(plan->h_OptionData));
    checkCudaErrors(hipFree(plan->d_CallValue));
    checkCudaErrors(hipFree(plan->d_OptionData));
}

//Main computations
extern "C" void MonteCarloGPU(TOptionPlan *plan, hipStream_t stream)
{
    __TOptionValue *h_CallValue = plan->h_CallValue;

    if (plan->optionCount <= 0 || plan->optionCount > MAX_OPTIONS)
    {
        printf("MonteCarloGPU(): bad option count.\n");
        return;
    }
    printf("Thread %d.\n",THREAD_N);

    __TOptionData * h_OptionData = (__TOptionData *)plan->h_OptionData;

    for (int i = 0; i < plan->optionCount; i++)
    {
        const double           T = plan->optionData[i].T;
        const double           R = plan->optionData[i].R;
        const double           V = plan->optionData[i].V;
        const double       MuByT = (R - 0.5 * V * V) * T;
        const double    VBySqrtT = V * sqrt(T);
        h_OptionData[i].S        = (real)plan->optionData[i].S;
        h_OptionData[i].X        = (real)plan->optionData[i].X;
        h_OptionData[i].MuByT    = (real)MuByT;
        h_OptionData[i].VBySqrtT = (real)VBySqrtT;
    }

    checkCudaErrors(hipMemcpyAsync(
                        plan->d_OptionData,
                        h_OptionData,
                        plan->optionCount * sizeof(__TOptionData),
                        hipMemcpyHostToDevice, stream
                    ));

    MonteCarloOneBlockPerOption <<<plan->gridSize, THREAD_N, 0, stream >>> (
        plan->rngStates,
        (__TOptionData *)(plan->d_OptionData),
        (__TOptionValue *)(plan->d_CallValue),
        plan->pathN,
        plan->optionCount
    );
    getLastCudaError("MonteCarloOneBlockPerOption() execution failed\n");


    checkCudaErrors(hipMemcpyAsync(
                        h_CallValue,
                        plan->d_CallValue,
                        plan->optionCount * sizeof(__TOptionValue), hipMemcpyDeviceToHost, stream
                    ));

    //hipDeviceSynchronize();
}

